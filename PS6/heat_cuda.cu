#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define cudaCheckErrors(msg)                                   \
    do                                                         \
    {                                                          \
        hipError_t __err = hipGetLastError();                \
        if (__err != hipSuccess)                              \
        {                                                      \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                    msg, hipGetErrorString(__err),            \
                    __FILE__, __LINE__);                       \
            fprintf(stderr, "*** FAILED - ABORTING\n");        \
            exit(1);                                           \
        }                                                      \
    } while (0)




/* Functions to be implemented: */
float ftcs_solver_gpu ( int step, int block_size_x, int block_size_y );
float ftcs_solver_gpu_shared ( int step, int block_size_x, int block_size_y );
float ftcs_solver_gpu_texture ( int step, int block_size_x, int block_size_y );
void external_heat_gpu ( int step, int block_size_x, int block_size_y );
void transfer_from_gpu( int step );
void transfer_to_gpu();
void device_allocation();

/* Prototypes for functions found at the end of this file */
void write_temp( int step );
void print_local_temps();
void init_temp_material();
void init_local_temp();
void host_allocation();
void add_time(float time);
void print_time_stats();

/*
 * Physical quantities:
 * k                    : thermal conductivity      [Watt / (meter Kelvin)]
 * rho                  : density                   [kg / meter^3]
 * cp                   : specific heat capacity    [kJ / (kg Kelvin)]
 * rho * cp             : volumetric heat capacity  [Joule / (meter^3 Kelvin)]
 * alpha = k / (rho*cp) : thermal diffusivity       [meter^2 / second]
 *
 * Mercury:
 * cp = 0.140, rho = 13506, k = 8.69
 * alpha = 8.69 / (0.140*13506) =~ 0.0619
 *
 * Copper:
 * cp = 0.385, rho = 8960, k = 401
 * alpha = 401.0 / (0.385 * 8960) =~ 0.120
 *
 * Tin:
 * cp = 0.227, k = 67, rho = 7300
 * alpha = 67.0 / (0.227 * 7300) =~ 0.040
 *
 * Aluminium:
 * cp = 0.897, rho = 2700, k = 237
 * alpha = 237 / (0.897 * 2700) =~ 0.098
 */

const float MERCURY = 0.0619;
const float COPPER = 0.116;
const float TIN = 0.040;
const float ALUMINIUM = 0.098;

/* Discretization: 5cm square cells, 2.5ms time intervals */
const float
    h  = 5e-2,
    dt = 2.5e-3;

/* Size of the computational grid - 1024x1024 square */
const int GRID_SIZE[2] = {2048, 2048};

/* Parameters of the simulation: how many steps, and when to cut off the heat */
const int NSTEPS = 10000;
const int CUTOFF = 5000;

/* How often to dump state to file (steps). */
const int SNAPSHOT = 500;


//
#define BLOCKY 8
#define BLOCKX 8


/* For time statistics */
float min_time = -2.0;
float max_time = -2.0;
float avg_time = 0.0;

/* Arrays for the simulation data, on host */
float
    *material,          // Material constants
    *temperature;       // Temperature field

/* Arrays for the simulation data, on device */
float
    *material_device,           // Material constants
    *temperature_device[2];      // Temperature field, 2 arrays 


/* Allocate arrays on GPU */
void device_allocation(){
    size_t total_grid_size =GRID_SIZE[0]*GRID_SIZE[1];
    hipMalloc(&material_device, total_grid_size* sizeof(float));
    hipMalloc(&temperature_device[0], total_grid_size* sizeof(float));
    hipMalloc(&temperature_device[1], total_grid_size* sizeof(float));
}

/* Transfer input to GPU */
void transfer_to_gpu(){
    hipMemcpy(material_device, material, GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(temperature_device[0], temperature, GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), hipMemcpyHostToDevice);
    // cudaMemcpy(temperature_device[1], temperature, GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), cudaMemcpyHostToDevice);
}

/* Transfer output from GPU to CPU */
void transfer_from_gpu(int step){
    // cudaMemcpy(material, material_device, GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), cudaMemcpyDeviceToHost);
    // cudaMemcpy(temperature, temperature_device[0] , GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), cudaMemcpyDeviceToHost);
    printf("2 value: %f \n", temperature[1]);
    hipMemcpy(temperature, temperature_device[(step+1)%2] , GRID_SIZE[0]*GRID_SIZE[1]*sizeof(float), hipMemcpyDeviceToHost);
    
    printf("3 value: %f \n", temperature[1]);

}

 // Plain/global memory only kernel
__global__ void  ftcs_kernel(int step, float *zero, float *one, float *material_device){ /* Add arguments here   */
    const int GRID_SIZE[2] = {2048, 2048};
    int i = (blockIdx.x * (2048/128)) + threadIdx.x;
    int j = (blockIdx.y * (2048/128)) + threadIdx.y;
    int palceInArray = i * 2048 + j;

    if (i == 0){
        i=1;
    } 

    if (j == 0){
        j=1;
    }

    if (i>= 2047){
        i = 2047-1;
    }

    if (j>= 2047){
        j = 2047-1;
    }

    if (step % 2 == 0){
        one[palceInArray] = zero[palceInArray] + material_device[palceInArray]*(
            zero[(i+1)*2048 + (j+0)] + 
            zero[(i-1)*2048 + (j+0)] +  
            zero[(i+0)*2048 + (j+1)] + 
            zero[(i+0)*2048 + (j-1)] - 
            4*zero[palceInArray]);
    } else {
        // zero[palceInArray] = one[palceInArray] + material_device[palceInArray]*(one[(i+1)*2048 + j] + one[(i-1)*2048 + j] +  one[(i)*2048 + j+1] + one[(i*2048) + (j-1)] - 4*one[palceInArray]);
         zero[palceInArray] = one[palceInArray] + material_device[palceInArray]*(
            one[(i+1)*2048 + (j+0)] + 
            one[(i-1)*2048 + (j+0)] +  
            one[(i+0)*2048 + (j+1)] + 
            one[(i+0)*2048 + (j-1)] - 
            4*one[palceInArray]);

    }
    // if (threadIdx.)  {
    //     printf("%d\n", blockIdx.x);
    // //     printf("i: %d, j: %d\n",i, j );
    // //     // printf("Hell/so from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    // //     // printf("out: %f  and in: %f \n", one[i,j], zero[i,j]);
    // // }
    // if (blockIdx.y % 20 == 0){
    //     printf("Y: %d\n",blockIdx.y );
    // }
}

/* Shared memory kernel */
__global__ void  ftcs_kernel_shared( /* Add arguments here */ ){
    
}

/* Texture memory kernel */
__global__ void  ftcs_kernel_texture( /* Add arguments here */ ){

}

/* External heat kernel, should do the same work as the external
 * heat function in the serial code 
 */
__global__ void external_heat_kernel( /* Add arguments here */ ){

}

/* Set up and call ftcs_kernel
 * should return the execution time of the kernel
 */

//Dele med mindre. 
float ftcs_solver_gpu( int step, int block_size_x, int block_size_y ){
     // Compute thread-block size
    dim3 gridBlock(block_size_x, block_size_y); 
    dim3 threadBlock(GRID_SIZE[0]/block_size_x, GRID_SIZE[1]/block_size_y);

    // Call kernel
    ftcs_kernel<<<gridBlock, threadBlock>>>( step, temperature_device[0], temperature_device[1], material_device);
    
    float time = -1.0;
    return time;
}

/* Set up and call ftcs_kernel_shared
 * should return the execution time of the kernel
 */
float ftcs_solver_gpu_shared( int step, int block_size_x, int block_size_y ){
    
    float time = -1.0;
    return time;
}

/* Set up and call ftcs_kernel_texture
 * should return the execution time of the kernel
 */
float ftcs_solver_gpu_texture( int step, int block_size_x, int block_size_y ){
    
    float time = -1.0;
    return time;
}


/* Set up and call external_heat_kernel */
void external_heat_gpu( int step, int block_size_x, int block_size_y ){
}

void print_gpu_info(){
  int n_devices;
  hipGetDeviceCount(&n_devices);
  printf("Number of CUDA devices: %d\n", n_devices);
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, 0);
  printf("CUDA device name: %s\n" , device_prop.name);
  printf("Compute capability: %d.%d\n", device_prop.major, device_prop.minor);
}


int main ( int argc, char **argv ){
    
    // Parse command line arguments
    int version = 0;
    int block_size_x = 0;
    int block_size_y = 0;
    if(argc != 4){
        printf("Useage: %s <version> <block_size_x> <block_size_y>\n\n<version> can be:\n0: plain\n1: shared memory\n2: texture memory\n", argv[0]);
        exit(0);
    }
    else{
        version = atoi(argv[1]);
        block_size_x = atoi(argv[2]);
        block_size_y = atoi(argv[3]);
    }
    
    print_gpu_info();
    
    // Allocate and initialize data on host
    host_allocation();

    init_temp_material();

    // Allocate arrays on device, and transfer inputs
    device_allocation();
    transfer_to_gpu();
        
    // Main integration loop
    for( int step=0; step<NSTEPS; step += 1 ){
        
        if( step < CUTOFF ){
            external_heat_gpu ( step, block_size_x, block_size_y );
        }
        
        float time;
        // Call selected version of ftcs slover
        if(version == 2){
            time = ftcs_solver_gpu_texture( step, block_size_x, block_size_y );
        }
        else if(version == 1){
            time = ftcs_solver_gpu_shared(step, block_size_x, block_size_y);
        }
        else{
            time = ftcs_solver_gpu(step, block_size_x, block_size_y);
        }
        
        add_time(time);
        
        if((step % SNAPSHOT) == 0){
            // Transfer output from device, and write to file
            transfer_from_gpu(step);
            write_temp(step);
        }
    }
    
    print_time_stats();
        
    exit ( EXIT_SUCCESS );
}


void host_allocation(){
    size_t temperature_size =GRID_SIZE[0]*GRID_SIZE[1];
    temperature = (float*) calloc(temperature_size, sizeof(float));
    size_t material_size = (GRID_SIZE[0])*(GRID_SIZE[1]); 
    material = (float*) calloc(material_size, sizeof(float));
}


void init_temp_material(){
    
    for(int x = 0; x < GRID_SIZE[0]; x++){
        for(int y = 0; y < GRID_SIZE[1]; y++){
            temperature[y * GRID_SIZE[0] + x] = 10.0;

        }
    }
    
    for(int x = 0; x < GRID_SIZE[0]; x++){
        for(int y = 0; y < GRID_SIZE[1]; y++){
            temperature[y * GRID_SIZE[0] + x] = 20.0;
            material[y * GRID_SIZE[0] + x] = MERCURY * (dt/(h*h));
        }
    }
    
    /* Set up the two blocks of copper and tin */
    for(int x=(5*GRID_SIZE[0]/8); x<(7*GRID_SIZE[0]/8); x++ ){
        for(int y=(GRID_SIZE[1]/8); y<(3*GRID_SIZE[1]/8); y++ ){
            material[y * GRID_SIZE[0] + x] = COPPER * (dt/(h*h));
            temperature[y * GRID_SIZE[0] + x] = 60.0;
        }
    }
    
    for(int x=(GRID_SIZE[0]/8); x<(GRID_SIZE[0]/2)-(GRID_SIZE[0]/8); x++ ){
        for(int y=(5*GRID_SIZE[1]/8); y<(7*GRID_SIZE[1]/8); y++ ){
            material[y * GRID_SIZE[0] + x] = TIN * (dt/(h*h));
            temperature[y * GRID_SIZE[0] + x] = 60.0;
        }
    }

    /* Set up the heating element in the middle */
    for(int x=(GRID_SIZE[0]/4); x<=(3*GRID_SIZE[0]/4); x++){
        for(int y=(GRID_SIZE[1]/2)-(GRID_SIZE[1]/16); y<=(GRID_SIZE[1]/2)+(GRID_SIZE[1]/16); y++){
            material[y * GRID_SIZE[0] + x] = ALUMINIUM * (dt/(h*h));
            temperature[y * GRID_SIZE[0] + x] = 100.0;
        }
    }
}


void add_time(float time){
    avg_time += time;
    
    if(time < min_time || min_time < -1.0){
        min_time = time;
    }
    
    if(time > max_time){
        max_time = time;
    }
}

void print_time_stats(){
    printf("Kernel execution time (min, max, avg): %f %f %f\n", min_time, max_time, avg_time/NSTEPS);
}

/* Save 24 - bits bmp file, buffer must be in bmp format: upside - down
 * Only works for images which dimensions are powers of two
 */
void savebmp(char *name, unsigned char *buffer, int x, int y) {
  FILE *f = fopen(name, "wb");
  if (!f) {
    printf("Error writing image to disk.\n");
    return;
  }
  unsigned int size = x * y * 3 + 54;
  unsigned char header[54] = {'B', 'M',
                      size&255,
                      (size >> 8)&255,
                      (size >> 16)&255,
                      size >> 24,
                      0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, x&255, x >> 8, 0,
                      0, y&255, y >> 8, 0, 0, 1, 0, 24, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  fwrite(header, 1, 54, f);
  fwrite(buffer, 1, GRID_SIZE[0] * GRID_SIZE[1] * 3, f);
  fclose(f);
}

void fancycolour(unsigned char *p, float temp) {
    
    if(temp <= 25){
        p[2] = 0;
        p[1] = (unsigned char)((temp/25)*255);
        p[0] = 255;
    }
    else if (temp <= 50){
        p[2] = 0;
        p[1] = 255;
        p[0] = 255 - (unsigned char)(((temp-25)/25) * 255);
    }
    else if (temp <= 75){
        
        p[2] = (unsigned char)(255* (temp-50)/25);
        p[1] = 255;
        p[0] = 0;
    }
    else{
        p[2] = 255;
        p[1] = 255 -(unsigned char)(255* (temp-75)/25) ;
        p[0] = 0;
    }
}

/* Create nice image from iteration counts. take care to create it upside down (bmp format) */
void output(char* filename){
    unsigned char *buffer = (unsigned char*)calloc(GRID_SIZE[0] * GRID_SIZE[1]* 3, 1);
    for (int j = 0; j < GRID_SIZE[1]; j++) {
        for (int i = 0; i < GRID_SIZE[0]; i++) {
        int p = ((GRID_SIZE[1] - j - 1) * GRID_SIZE[0] + i) * 3;
        fancycolour(buffer + p, temperature[j*GRID_SIZE[0] + i]);
      }
    }
    /* write image to disk */
    savebmp(filename, buffer, GRID_SIZE[0], GRID_SIZE[1]);
    free(buffer);
}


void write_temp (int step ){
    char filename[15];
    sprintf ( filename, "data/%.4d.bmp", step/SNAPSHOT );

    output ( filename );
    printf ( "Snapshot at step %d\n", step );
    printf("%s\n", hipGetErrorString(hipGetLastError()));
}
